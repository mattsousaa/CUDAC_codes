
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <stdio.h>

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy*nx + ix;
    if(ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if(ix < nx){
        for(int iy = 0; iy < ny; iy++){
            int idx = iy*nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny){
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    unsigned int idx = iy*nx + ix;
    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}